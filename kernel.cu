#include "hip/hip_runtime.h"
#include <stdio.h>
#include <fstream>
#include <iostream>
#include <time.h>
#include "hip/hip_runtime.h"
#include ""

using namespace std;


const int DATA_SET_SIZE = 100000;
const int MOVES = 3;
const int PERMUTATIONS = MOVES * MOVES;
int NAB[PERMUTATIONS];
int dataSet[DATA_SET_SIZE];
const int threadsPerBlock = 256;
const int blocksPerGrid = (DATA_SET_SIZE + threadsPerBlock - 1) / threadsPerBlock;


// Generate a random set of integers each being in the range 0 to MOVES - 1
// Save numbers to file to ensure tests can be repeated.
void GenerateData() {
	ofstream out("data.dat", ios::out | ios::binary);
	for (int n = 0; n < DATA_SET_SIZE; n++) {
		int i = rand() % MOVES;
		out.write((char*)&i, sizeof(i));
	}
	out.close();
}

// I've found the initialise function isn't used in the rest of the code.

//void InitialiseNAB() { 
//	for (int n = 0; n < PERMUTATIONS; n++) {
//		NAB[n] = 0;
//	}
//} 


// Populate data array with contents of file
void GetData() {
	ifstream in("data.dat", ios::in | ios::binary);
	for (int n = 0; n < DATA_SET_SIZE; n++) {
		in.read((char*)&dataSet[n], sizeof(int));
	}
	in.close();
}


__device__ int GetIndex(int firstMove, int secondMove) { //device allows function to be called by global functions
		//if (firstMove == 0 && secondMove == 0) return 0;
		//if (firstMove == 0 && secondMove == 1) return 1;
		//if (firstMove == 0 && secondMove == 2) return 2;
		//if (firstMove == 1 && secondMove == 0) return 3;
		//if (firstMove == 1 && secondMove == 1) return 4;
		//if (firstMove == 1 && secondMove == 2) return 5;
		//if (firstMove == 2 && secondMove == 0) return 6;
		//if (firstMove == 2 && secondMove == 1) return 7;
		//if (firstMove == 2 && secondMove == 2) return 8;
	/*if (firstMove == 0) return firstMove + secondMove;
	if (firstMove == 1) return firstMove * MOVES + secondMove;
	if (firstMove == 2) return firstMove * MOVES + secondMove;*/ //atempt to shorten the calcluation
	return firstMove * MOVES + secondMove; //simplified into 1 calculation to make the comparison run faster. tested the calculation and it has the same return based on the int values.
	
}

void DisplayNAB() {
	int check = 0;
	cout << endl;
	for (int n = 0; n < PERMUTATIONS; n++) {
		cout << "Index " << n << " : " << NAB[n] << endl;
		check += NAB[n];
	}
	// Total should be one less than DATA_SET_SIZE as first value doesn't have a previous value to compare.
	cout << "Total : " << check << endl;
}




__global__ void PopulateNAB(int* dataSet, int* NAB, float* totals) {
	__shared__ int cache[threadsPerBlock];

	float data = 0;
	int tid = blockIdx.x * blockDim.x + threadIdx.x; //input array offset
	int cacheIndex = threadIdx.x;
	cache[cacheIndex] = data;
	__syncthreads(); //sync function

	int i = blockDim.x / 2; //reduction kernal function
	while (i != 0) {
		if (cacheIndex < i) {
			cache[cacheIndex] += cache[cacheIndex + i];
		}
		__syncthreads();
		i /= 2;
	}
	if (cacheIndex == 0) {
		totals[blockIdx.x] - cache[0];
	}

	int index;
	int previous = dataSet[0];
	while(tid < DATA_SET_SIZE) { // main function
		index = GetIndex(previous, dataSet[tid]);
		atomicAdd(&NAB[index], 1); //addition of point values
		previous = dataSet[tid];
		data += dataSet[tid] * NAB[tid];
		tid += blockDim.x * gridDim.x;
	}

}

int main() {
	hipEvent_t start, stop; //creating the timer.
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);



	srand(time(NULL));
	GenerateData();
	GetData();
	int* dev_dataSet; //initialise pointers
	int* dev_NAB;
	float* dev_totals;

	hipMalloc((void**)&dev_dataSet, DATA_SET_SIZE * sizeof(int)); //allocate memory
	hipMalloc((void**)&dev_NAB, PERMUTATIONS * sizeof(int));
	hipMalloc((void**)&dev_totals, sizeof(float) * blocksPerGrid);
	hipMemcpy(dev_dataSet, dataSet, DATA_SET_SIZE * sizeof(int), hipMemcpyHostToDevice);
	hipMemcpy(dev_NAB, NAB, PERMUTATIONS * sizeof(int), hipMemcpyHostToDevice);

	const int size = 100; // executes on 100 block. uses muliple processors on the gpu

	PopulateNAB << <blocksPerGrid, threadsPerBlock >> > (dev_dataSet, dev_NAB, dev_totals); //kernel function
	hipMemcpy(NAB, dev_NAB, PERMUTATIONS * sizeof(int), hipMemcpyDeviceToHost); //copies data
	hipFree(dev_dataSet); //free up the memory.
	hipFree(dev_NAB);
	hipFree(dev_totals);
	//PopulateNAB();
	DisplayNAB();
	
	
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop); //stops the timer
	float elapseTime;
	hipEventElapsedTime(&elapseTime, start, stop); //records the time between start and stop
	printf("Time to generate: %3.1f ms \n", elapseTime); 
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}
